#include <stdio.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 10

int data[DATA_SIZE];

__global__ static void sum(int *num, int* result,int *result1)
{
    int sum = 0;
    int i;
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = sum;
    *result1= 1024;
}


bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}


void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = i;
    }
}

int main()
{
    
    if(!InitCUDA()) {
        return 0;
    }
    
    printf("CUDA initialized.\n");

    GenerateNumbers(data, DATA_SIZE);
    for(int i = 0; i < DATA_SIZE; i++) {
        printf("data[%d]=%d\n",i,data[i]);
    }
    int *gpudata, *result,*result1;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMalloc((void**) &result1, sizeof(int));
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,hipMemcpyHostToDevice);


    sum<<<1, 1>>>(gpudata, result,result1);

    int *sum;
    int *results;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&results, result1, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(result1);

    printf("sum: %d\n", sum);
    printf("result1: %d\n",results);


    int sums = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        
	
        sums += data[i] * data[i];
    }
    printf("sum (CPU): %d\n", sums);

    return 0;
}
