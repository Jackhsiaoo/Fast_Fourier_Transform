#include <stdio.h>
#include <hip/hip_runtime.h>

int fft(double *x_r, double *x_i, double *y_r, double *y_i, int N);


int main()
{
	
	
	
	
	return 0;
}


int fft(double *x_r, double *x_i, double *y_r, double *y_i, int N){
	int n,p, m=0;
	int N0,M0;
	int order[100];
	
	//termination conditions
	if(N==1){
		
		y_r[0] = x_r[0];
		y_i[0] = x_i[0];
		
		return 0;
	}
	
	
	
	
	for(n=0;n<N;++n)
	{
		y_r[n] = x_r[n];
		y_i[n] = x_i[n];
	}
	
	N0=N;
	p=1;
	while (N0>1) {
		if ((N0%2)==0) {
			p=2;
		}else if ((N0%3)==0){
			p=3;
		}else if ((N0%5)==0){
			p=5;
		}else{
			p=1;
		}
		M0=0;
		while (M0<N) {
			
			groupn(y_r+M0, y_i+M0, N0, p);
			M0+=N0;
		}
		order[m]=p;
		m++;
		N0/=p;
	}
	
	
	
	while (N0<N) {
		m--;
		butterfly(y_r, y_i, N, order[m],N0);
		
		
		N0*=order[m];
	}
	
	
	return 0;
}
